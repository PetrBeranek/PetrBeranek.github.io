#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <GL/glut.h>
#include <pthread.h>

#include <windows.h>

#include "Vec3.h"
#include "Bitmap.h"
#include "RayTask.h"
#include "Collidable.h"
#include "Sphere.h"
#include "Point.h"
#include "Triangle.h"
#include "Col.h"
#include "Camera.h"
#include "Structures.h"
#include "RayBuffer.h"
#include "PointLight.h"
#include "Material.h"
#include "KDTree.h"

#define NOMINMAX

///image resolution
#define IMAGE_WIDTH  500 
///image resolution
#define IMAGE_HEIGHT 500 

///input from user
int INPUT_METHOD;
///input from user
int INPUT_TRAVERSING;
///input from user
int INPUT_SCENE;	

RayBuffer buffer;

Bitmap *GLOBAL_BITMAP = NULL;
float *GLOBAL_PIXEL_BUFFER;
RayCompact *GLOBAL_RAYBUFFER;
///main camera
PinholeCamera cam;			

///compacted KDTree
__device__ KDTreeNodeCompact *CUDA_Nodes;
///compacted Spheres
__device__ SphereCompact *CUDA_Spheres;
///compacted Triangles
__device__ TriangleCompact *CUDA_Triangles;
///compacted  Materials
__device__ MaterialCompact *CUDA_Materials;
///compacted Lights
__device__ PointLightCompact *CUDA_Lights;
///compacted Rays
__device__ RayCompact *CUDA_Rays;
///compacted Triangle Indices
__device__ int *CUDA_Indices_Tri;
///compacted Sphere Indices
__device__ int *CUDA_Indices_Sph;
///output Color Buffer
__device__ float *CUDA_ColorBuffer;


void testBitmap();

/**
Normalize float[3] vector
*/
void normalize(float3 &vec)
{
	float m = sqrt((vec.x*vec.x) + (vec.y*vec.y) + (vec.z*vec.z));
}
/**
Get current time.
*/
long long getTime()
{
    SYSTEMTIME t;
    GetSystemTime(&t);

	long long time = ((((((t.wHour * 60) + t.wMinute) * 60) + t.wSecond) * 1000) + t.wMilliseconds);
	return time;
}
/**
Draw GLOBAL_BITMAP to glut window
*/
void DisplayFunc( void ) {
  glClearColor( 0.0, 1.0, 0.0, 1.0 );
  glClear( GL_COLOR_BUFFER_BIT );
  
  //testBitmap();

  if(GLOBAL_BITMAP != NULL ){
	glDrawPixels( GLOBAL_BITMAP->width, GLOBAL_BITMAP->height, GL_RGBA, GL_UNSIGNED_BYTE, GLOBAL_BITMAP->pixels );
  }

  glutSwapBuffers();
}
/**
Initialize Light collection
*/
void initializeLight()
{
	GLOBAL_BUFFER_LIGHT = LightCol();
}
/**
Initialize bitmap
*/
void initializeBitmap()
{
	GLOBAL_BITMAP = new Bitmap(IMAGE_WIDTH, IMAGE_HEIGHT);
	printf("initialized\n");
}
/**
print compactized KDTree
*/
void printCompactTree()
{
	CompactTree *c = &GLOBAL_COMPACT;
	KDTreeNodeCompact *n = (c->nodes);
	SphereCompact *s = (c->sphere);
	TriangleCompact *t = (c->tri);
	//printf("TRIANGLE BUFFER ADDRESS %d \n",c->tri);
	MaterialCompact *m = (c->mat);
	printf("NODES \n");
	for(int i = 0; i < c->countNode; i++)
	{
		printf("id %d   %d <--> %d %d <--> %d %d <--> %d     s %d t %d     %d \n", n[i].id, n[i].neighbors[0], n[i].neighbors[1], n[i].neighbors[2], n[i].neighbors[3], n[i].neighbors[4], n[i].neighbors[5], n[i].countSph, n[i].countTri, (n[i].isLeaf)?0:1);
		printf("sph from %d next %d tri from %d to %d\n",n[i].offsetSph, n[i].countSph,n[i].offsetTri,n[i].countTri);
	}
	printf("SPHERE INDICES\n");
	for(int i = 0; i < c->sphereIndicesSize; i++)
	{
		printf("%d ",c->sphereIndices[i]);
	}
	printf("SPHERES\n");
	for(int i = 0; i < c->countSph; i++)
	{
		printf("id %d  %f %f %f    %f    mat %d \n", i, s[i].pos[0], s[i].pos[1], s[i].pos[2], s[i].r, s[i].mat);
	}
	printf("TRIANGLE INDICES\n");
	for(int i = 0; i < c->triangleIndicesSize; i++)
	{
		printf("%d ",c->trinagleIndices[i]);
	}
	printf("TRIANGLES\n");
	for(int i = 0; i < c->countTri; i++)
	{
		printf("id %d  %f %f %f   %f %f %f   %f %f %f    mat %d \n", i,t[i].v0[0],t[i].v0[1],t[i].v0[2],t[i].v1[0],t[i].v1[1],t[i].v1[2],t[i].v2[0],t[i].v2[1],t[i].v2[2], s[i].mat);
	}
	printf("MATERIALS\n");
	for(int i = 0; i < c->countMat; i++)
	{
		printf("id %d   %f %f %f \n",i,m[i].diff[0],m[i].diff[1],m[i].diff[2]);
	}
	printf("\nEND\n");

}
/**
test scene
*/
void buildTestBall()
{
	float w,h,f;
	w = 6.0f;
	h = 6.0f;
	f = 4.0f;

	cam = PinholeCamera(Vec3(0.0f,0.0f,0.0f), Vec3(0.0f,0.0f,-5.0f), Vec3(0.0f,1.0f,0.0f),f,w,h);
	buffer = RayBuffer();
	PointLight *l;
	l = new PointLight(Vec3(0.0f,4.0f,0.0f),Col(1.0f,1.0f,1.0f), 50, 1, 1, 1);
	GLOBAL_BUFFER_LIGHT.addLight(*l);
	//l = new PointLight(Vec3(0.0f,-1.0f,-1.0f),Col(1.0f,1.0f,1.0f), 3, 1, 1, 1);
	//GLOBAL_BUFFER_LIGHT.addLight(*l);
	


	std::vector<Collidable *> *vec = new std::vector<Collidable *>();

	Collidable *c;
	
	c = new Sphere(Vec3(-0.750f, 0.0f, -8.0f), 1.0f, Material::getMaterialIndex(1));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(1.250f, 0.0f, -8.0f), 1.0f, Material::getMaterialIndex(2));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(0.0f, 0.0f, -2.0f), 0.30f, Material::getMaterialIndex(3));
	c->precomputeBounds();
	vec->push_back(c);
	

	GLOBAL_TREE.buildTreeStart(*vec);
}
/**
test scene
*/
void buildMassiveScene1()
{
	float w,h,f;
	w = 6.0f;
	h = 6.0f;
	f = 4.0f;

	cam = PinholeCamera(Vec3(0.0f,0.0f,0.0f), Vec3(0.0f,0.0f,-5.0f), Vec3(0.0f,1.0f,0.0f),f,w,h);
	buffer = RayBuffer();
	PointLight *l;
	l = new PointLight(Vec3(0.0f,4.0f,0.0f),Col(1.0f,1.0f,1.0f), 50, 1, 1, 1);
	GLOBAL_BUFFER_LIGHT.addLight(*l);

	std::vector<Collidable *> *vec = new std::vector<Collidable *>();

	Collidable *c;
	int cons = 20;
	float off = 01.0f;
	for(int i = 0; i < cons; i++)
		for(int j = 0; j < cons; j++)
			for(int k = 0; k < cons; k++)
			{
				c = new Sphere(Vec3(-off*cons/2.0f + off * i, -off*cons/2.0f + off * j, -5.0f - k*off), 0.250f, Material::getMaterialIndex(2));
				c->precomputeBounds();
				vec->push_back(c);
			}
	
	GLOBAL_TREE.buildTreeStart(*vec);
}

void buildMassiveScene2()
{
	float w,h,f;
	w = 6.0f;
	h = 6.0f;
	f = 4.0f;

	cam = PinholeCamera(Vec3(0.0f,0.0f,0.0f), Vec3(0.0f,0.0f,-5.0f), Vec3(0.0f,1.0f,0.0f),f,w,h);
	buffer = RayBuffer();
	PointLight *l;
	l = new PointLight(Vec3(0.0f,0.0f,-8.0f),Col(1.0f,1.0f,1.0f), 50, 1, 1, 1);
	GLOBAL_BUFFER_LIGHT.addLight(*l);
	l = new PointLight(Vec3(0.0f,0.0f,0.0f),Col(1.0f,1.0f,1.0f), 150, 1, 1, 1);
	GLOBAL_BUFFER_LIGHT.addLight(*l);

	


	std::vector<Collidable *> *vec = new std::vector<Collidable *>();
	Collidable *c;
	float s = 0.250f;
	float r = 0.15f;
	float g = 0.025;
	int d = 30;
	float spin = 3.0f;
	float b = 3.14f/spin;
	float a = 3.14f/(15.0f);
	float bou = 3.14f/(4.0f);

	float tmp;
	float baseD = 6.0f;

	
	c = new Sphere(Vec3( 0, 0, -baseD ), 0.250f, Material::getMaterialIndex(1));
	c->precomputeBounds();
	vec->push_back(c);
	for(int j = 0; j < spin; j++)
	{
		tmp = j*b;
		for(int i = 1; i <= d; i++)
		{
			c = new Sphere(Vec3( s * i * sin(tmp + a*i), s * i * cos(tmp + a*i), -baseD - 2*sin(0.5f * bou * i) ), r , Material::getMaterialIndex(11));
			c->precomputeBounds();
			vec->push_back(c);
			c = new Sphere(Vec3( -s * i * sin(tmp + a*i), -s * i * cos(tmp + a*i), -baseD - 2*sin(0.5f * bou * i) ), r , Material::getMaterialIndex(11));
			c->precomputeBounds();
			vec->push_back(c);
		}
	}
	for(int j = 0; j < spin; j++)
	{
		tmp = j*b;
		for(int i = 1; i <= d; i++)
		{
			c = new Sphere(Vec3( -s * i * sin(tmp + a*i), s * i * cos(tmp + a*i), -baseD - 2*sin(0.5f * bou * i) ), r , Material::getMaterialIndex(12));
			c->precomputeBounds();
			vec->push_back(c);
			c = new Sphere(Vec3( s * i * sin(tmp + a*i), -s * i * cos(tmp + a*i), -baseD - 2*sin(0.5f * bou * i) ), r , Material::getMaterialIndex(12));
			c->precomputeBounds();
			vec->push_back(c);
		}
	}
	
	GLOBAL_TREE.buildTreeStart(*vec);
}

void buildMassiveScene3()
{
	float w,h,f;
	w = 6.0f;
	h = 6.0f;
	f = 4.0f;

	cam = PinholeCamera(Vec3(0.0f,0.0f,0.0f), Vec3(0.0f,0.0f,-5.0f), Vec3(0.0f,1.0f,0.0f),f,w,h);
	buffer = RayBuffer();
	PointLight *l;
	l = new PointLight(Vec3(0.0f,2.0f,0.0f),Col(1.0f,1.0f,1.0f), 20, 1, 1, 1);
	GLOBAL_BUFFER_LIGHT.addLight(*l);
	l = new PointLight(Vec3(1.0f,-2.0f,0.0f),Col(1.0f,1.0f,1.0f), 10, 1, 1, 1);
	GLOBAL_BUFFER_LIGHT.addLight(*l);

	


	std::vector<Collidable *> *vec = new std::vector<Collidable *>();
	Collidable *c;

	int con = 10;
	
	c = new Sphere(Vec3( 0.0f, 0.0f, 0.0f ), 01.0f, Material::getMaterialIndex(15));
	c->precomputeBounds();
	vec->push_back(c);

	for(int i = 1; i < con; i++)
	{
		c = new Sphere(Vec3( 0.3f * i, 0, -2-2*i ), 01.0f, Material::getMaterialIndex(15));
		c->precomputeBounds();
		vec->push_back(c);
	}
	
	GLOBAL_TREE.buildTreeStart(*vec);
}
/**
test scene
*/
void buildTestScene1()
{
	float w,h,f;
	w = 9.0f;
	h = 9.0f;
	f = 6.0f;

	cam = PinholeCamera(Vec3(0.0f,0.0f,0.0f), Vec3(0.0f,0.0f,-5.0f), Vec3(0.0f,1.0f,0.0f),f,w,h);
	buffer = RayBuffer();
	PointLight *l;
	l = new PointLight(Vec3(1.0f,0.0f,1.0f),Col(1.0f,1.0f,1.0f), 15, 1, 1, 1);
	GLOBAL_BUFFER_LIGHT.addLight(*l);
	l = new PointLight(Vec3(-3.0f,3.0f,-1.0f),Col(1.0f,1.0f,1.0f), 10, 1, 1, 1);
	GLOBAL_BUFFER_LIGHT.addLight(*l);
	l = new PointLight(Vec3(2.0f,0.0f,0.0f),Col(0.30f,0.30f,0.30f), 4, 1, 1, 1);
	GLOBAL_BUFFER_LIGHT.addLight(*l);
	


	std::vector<Collidable *> *vec = new std::vector<Collidable *>();

	Collidable *c;

	c = new Sphere(Vec3(0.0f, .5f, -4.20f), 1.0f, Material::getMaterialIndex(1));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(-0.0f, -1.0f, -3.0f), 0.5f, Material::getMaterialIndex(2));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(-0.0f, -0.0f, -03.0f), 0.21f, Material::getMaterialIndex(3));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(0.7f, 0.30f, -2.5f), 0.5f, Material::getMaterialIndex(4));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(02.0f, 0.00f, -5.0f), 0.70f, Material::getMaterialIndex(4));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(0.9f, 0.00f, -02.50f), 0.50f, Material::getMaterialIndex(7));
	c->precomputeBounds();
	vec->push_back(c);
	

	GLOBAL_TREE.buildTreeStart(*vec);
	/*
	
	GLOBAL_TREE.isLeaf = true;
	GLOBAL_TREE.items = vec;
	GLOBAL_TREE.bounds[0] = -100;
	GLOBAL_TREE.bounds[1] = 100;
	GLOBAL_TREE.bounds[2] = -100;
	GLOBAL_TREE.bounds[3] = 100;
	GLOBAL_TREE.bounds[4] = -100;
	GLOBAL_TREE.bounds[5] = 100;

	GLOBAL_TREE.axis = -1;
	GLOBAL_TREE.depth = 0;
	GLOBAL_TREE.id = 55;
	*/
}
/**
test scene
*/
void buildTestScene2()
{
	float w,h,f;
	w = 9.0f;
	h = 9.0f;
	f = 4.0f;
	
	float con = 2;

	cam = PinholeCamera(Vec3(-0.0f,1.0f,0.0f), Vec3(-0.0f,0.0f,-5.0f), Vec3(0.0f,1.0f,0.0f),f,w,h);
	buffer = RayBuffer();
	PointLight *l;
	l = new PointLight(Vec3(0.0f, 01.0f * (con-0.10f), -con),Col(1.0f,1.0f,1.0f), 15, 1, 1, 1);
	GLOBAL_BUFFER_LIGHT.addLight(*l);
	l = new PointLight(Vec3(0.0f, 0.0f, 0.0f),Col(1.0f,1.0f,1.0f), 2, 1, 1, 0);
	//lights.addLight(*l);
	

	std::vector<Collidable *> *vec = new std::vector<Collidable *>();

	Collidable *c;
	
	//c = new Sphere(Vec3(0.0f, 0.0f, -1.0f), 0.50f, Material::getMaterialIndex(3));
	//c->precomputeBounds();
	//vec->push_back(c);

	c = new Sphere(Vec3(-1.0f, -1.0f, -3.0f), 1.0f, Material::getMaterialIndex(1));
	c->precomputeBounds();
	vec->push_back(c);


	//left
	c = new Triangle( Vec3(-con,-con,con), Vec3(-con,-con,-2*con), Vec3(-con,con,con), Material::getMaterialIndex(8));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Triangle( Vec3(-con,-con,-2*con), Vec3(-con,con,-2*con), Vec3(-con,con,con), Material::getMaterialIndex(8));
	c->precomputeBounds();
	vec->push_back(c);
	//right
	c = new Triangle( Vec3(con,-con,con), Vec3(con,con,con), Vec3(con,-con,-2*con), Material::getMaterialIndex(9));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Triangle( Vec3(con,-con,-2*con), Vec3(con,con,con), Vec3(con,con,-2*con), Material::getMaterialIndex(9));
	c->precomputeBounds();
	vec->push_back(c);
	//back
	c = new Triangle( Vec3(-con,con,-2*con), Vec3(-con,-con,-2*con), Vec3(con,-con,-2*con), Material::getMaterialIndex(10));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Triangle( Vec3(-con,con,-2*con), Vec3(con,-con,-2*con), Vec3(con,con,-2*con), Material::getMaterialIndex(10));
	c->precomputeBounds();
	vec->push_back(c);
	//bottom
	c = new Triangle( Vec3(-con,-con,con), Vec3(con,-con,con), Vec3(con,-con,-2*con), Material::getMaterialIndex(10));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Triangle( Vec3(-con,-con,-2*con), Vec3(-con,-con,con), Vec3(con,-con,-2*con), Material::getMaterialIndex(10));
	c->precomputeBounds();
	vec->push_back(c);
	//top
	c = new Triangle( Vec3(-con,con,con), Vec3(con,con,-2*con), Vec3(con,con,con), Material::getMaterialIndex(10));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Triangle( Vec3(-con,con,-2*con), Vec3(con,con,-2*con), Vec3(-con,con,con), Material::getMaterialIndex(10));
	c->precomputeBounds();
	vec->push_back(c);

	
	//front
	c = new Triangle( Vec3(-con,con,con), Vec3(con,-con,con), Vec3(-con,-con,con), Material::getMaterialIndex(2));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Triangle( Vec3(-con,con,con), Vec3(con,con,con), Vec3(con,-con,con), Material::getMaterialIndex(2));
	c->precomputeBounds();
	vec->push_back(c);
	

	GLOBAL_TREE.buildTreeStart(*vec);
}
/**
test scene
*/
void buildTestScene3()
{
	
	float w,h,f;
	w = 6.0f;
	h = 6.0f;
	f = 4.0f;
	
	
	//cam = PinholeCamera(Vec3(-3.0f,-3.0f,-4.0f), Vec3(-2.0f,-2.0f,-5.0f), Vec3(0.0f,1.0f,0.0f),f,w,h);
	cam = PinholeCamera(Vec3(0.0f,0.0f,1.0f), Vec3(0.0f,0.0f,-6.0f), Vec3(0.0f,1.0f,0.0f),f,w,h);
	buffer = RayBuffer();
	PointLight *l;
	l = new PointLight(Vec3(0.0f, 0.0f, 0.0f),Col(1.0f,1.0f,1.0f), 1, 1, 0, 0);
	GLOBAL_BUFFER_LIGHT.addLight(*l);
	//l = new PointLight(Vec3(0.0f, 0.0f, 0.0f),Col(1.0f,1.0f,1.0f), 2, 1, 1, 0);
	//lights.addLight(*l);
	
	
	std::vector<Collidable *> *vec1 = new std::vector<Collidable *>();
	std::vector<Collidable *> *vec2 = new std::vector<Collidable *>();
	std::vector<Collidable *> *vec = new std::vector<Collidable *>();

	Collidable *c;
	
	c = new Sphere(Vec3(-2.0f, -2.0f, -5.0f), 0.50f, Material::getMaterialIndex(11));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(-2.0f, -2.0f, -7.0f), 0.50f, Material::getMaterialIndex(12));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(-2.0f, -2.0f, -9.0f), 0.50f, Material::getMaterialIndex(13));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(-2.0f, -2.0f, -11.0f), 0.50f, Material::getMaterialIndex(14));
	c->precomputeBounds();
	vec->push_back(c);
	
	c = new Sphere(Vec3(2.0f, -2.0f, -5.0f), 0.50f, Material::getMaterialIndex(11));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(2.0f, -2.0f, -7.0f), 0.50f, Material::getMaterialIndex(12));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(2.0f, -2.0f, -9.0f), 0.50f, Material::getMaterialIndex(13));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(2.0f, -2.0f, -11.0f), 0.50f, Material::getMaterialIndex(14));
	c->precomputeBounds();
	vec->push_back(c);
	

	
	c = new Sphere(Vec3(-2.0f, 2.0f, -5.0f), 0.50f, Material::getMaterialIndex(11));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(-2.0f, 2.0f, -7.0f), 0.50f, Material::getMaterialIndex(12));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(-2.0f, 2.0f, -9.0f), 0.50f, Material::getMaterialIndex(13));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(-2.0f, 2.0f, -11.0f), 0.50f, Material::getMaterialIndex(14));
	c->precomputeBounds();
	vec->push_back(c);
	
	c = new Sphere(Vec3(2.0f, 2.0f, -5.0f), 0.50f, Material::getMaterialIndex(11));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(2.0f, 2.0f, -7.0f), 0.50f, Material::getMaterialIndex(12));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(2.0f, 2.0f, -9.0f), 0.50f, Material::getMaterialIndex(13));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Sphere(Vec3(2.0f, 2.0f, -11.0f), 0.50f, Material::getMaterialIndex(14));
	c->precomputeBounds();
	vec->push_back(c);

	GLOBAL_TREE.buildTreeStart(*vec);
}
/**
test scene
*/
void buildTestScene4()
{
	
	float w,h,f;
	w = 9.0f;
	h = 6.0f;
	f = 6.0f;
	
	
	//cam = PinholeCamera(Vec3(-3.0f,-3.0f,-4.0f), Vec3(-2.0f,-2.0f,-5.0f), Vec3(0.0f,1.0f,0.0f),f,w,h);
	cam = PinholeCamera(Vec3(0.0f,0.0f,-0.0f), Vec3(0.0f,0.0f,-6.0f), Vec3(0.0f,1.0f,0.0f),f,w,h);
	buffer = RayBuffer();
	PointLight *l;
	l = new PointLight(Vec3(0.0f, 0.0f, 0.0f),Col(1.0f,1.0f,1.0f), 1, 1, 0, 0);
	GLOBAL_BUFFER_LIGHT.addLight(*l);
	//l = new PointLight(Vec3(0.0f, 0.0f, 0.0f),Col(1.0f,1.0f,1.0f), 2, 1, 1, 0);
	//lights.addLight(*l);
	
	std::vector<Collidable *> *vec = new std::vector<Collidable *>();

	Collidable *c;
	
	float posX, posY, posZ;
	float t0x, t0y, t1x, t1y, t2x, t2y;
	t0x = -0.5f;
	t0y = -0.5f;
	t1x = 0.5f;
	t1y = -0.5f;
	t2x = -0.0f;
	t2y = 1.5f;

	posX = 0;
	posY = 0;
	posZ = -2;
	
	//c = new Triangle( Vec3(-1,-1,-3), Vec3(1,-1,-3), Vec3(0,1.5f,-3), Material::getMaterialIndex(1));
	
	//c = new Triangle( Vec3(posX+t0x,posY+t0y,posZ), Vec3(posX+t1x,posY+t1y,posZ), Vec3(posX+t2x,posY+t2y,posZ), Material::getMaterialIndex(11));
	//c->precomputeBounds();
	//vec->push_back(c);

	//c = new Sphere(Vec3(posX, posY, posZ), 0.50f, Material::getMaterialIndex(14));
	//c->precomputeBounds();
	//vec->push_back(c);

	posX = 2;
	//c = new Triangle( Vec3(posX+t0x,posY+t0y,posZ), Vec3(posX+t1x,posY+t1y,posZ), Vec3(posX+t2x,posY+t2y,posZ), Material::getMaterialIndex(11));
	//c->precomputeBounds();
	//vec->push_back(c);
	
	//c = new Sphere(Vec3(posX, posY, posZ), 0.50f, Material::getMaterialIndex(14));
	//c->precomputeBounds();
	//vec->push_back(c);

	c = new Triangle(Vec3(-5, -2, -5),Vec3(5, -2, -5),Vec3(0, 4, -5), Material::getMaterialIndex(14));
	c->precomputeBounds();
	vec->push_back(c);

	GLOBAL_TREE.buildTreeStart(*vec);
}
/**
test scene
*/
void buildTestScene5()
{
	float w,h,f;
	w = 9.0f;
	h = 9.0f;
	f = 4.0f;
	
	float con = 2;

	cam = PinholeCamera(Vec3(-0.0f,0.0f,0.0f), Vec3(-0.0f,0.0f,-5.0f), Vec3(0.0f,1.0f,0.0f),f,w,h);
	buffer = RayBuffer();
	PointLight *l;
	l = new PointLight(Vec3(0.0f, 01.0f * (con-0.10f), -con),Col(1.0f,1.0f,1.0f), 15, 1, 1, 1);
	GLOBAL_BUFFER_LIGHT.addLight(*l);
	l = new PointLight(Vec3(0.0f, 0.0f, 0.0f),Col(1.0f,1.0f,1.0f), 2, 1, 1, 0);
	//lights.addLight(*l);
	

	std::vector<Collidable *> *vec = new std::vector<Collidable *>();

	Collidable *c;

	Vec3 corner = Vec3(-6,-6,-8);
	Vec3 locX = Vec3(12,0,0);
	Vec3 locY = Vec3(0,12,0);
	int sizeX = 32;
	int sizeY = 32;
	/*
	c = new Triangle( Vec3(corner.x,corner.y,corner.z), Vec3(corner.x + 8,corner.y + 8,corner.z + 0), Vec3(corner.x ,corner.y + 8,corner.z), Material::getMaterialIndex(2));
	c->precomputeBounds();
	vec->push_back(c);
	c = new Triangle( Vec3(corner.x,corner.y,corner.z), Vec3(corner.x + 8,corner.y,corner.z), Vec3(corner.x + 8,corner.y + 8,corner.z + 0), Material::getMaterialIndex(1));
	c->precomputeBounds();
	vec->push_back(c);
	*/

	
	locX.mulLoc(1.0f/sizeX);
	locY.mulLoc(1.0f/sizeY);
	
	Vec3 tC;
	for(int i = 0; i < sizeX; i++)
	{
		for(int j = 0; j < sizeY; j++)
		{
			tC = Vec3(corner.x + i*locX.x + j*locY.x, corner.y + i*locX.y + j*locY.y, corner.z + i*locX.z + j*locY.z);

			
			c = new Triangle( Vec3(tC.x,tC.y,tC.z), Vec3(tC.x + locX.x + locY.x,tC.y + locX.y + locY.y,tC.z + locX.z + locY.z), Vec3(tC.x + locY.x ,tC.y + locY.y,tC.z + locY.z), Material::getMaterialIndex(2));
			c->precomputeBounds();
			vec->push_back(c);
			c = new Triangle( Vec3(tC.x,tC.y,tC.z), Vec3(tC.x + locX.x,tC.y + locX.y,tC.z + locX.z), Vec3(tC.x + locX.x + locY.x,tC.y + locX.y + locY.y,tC.z + locX.z + locY.z), Material::getMaterialIndex(1));
			c->precomputeBounds();
			vec->push_back(c);
		}
	}
	
	GLOBAL_TREE.buildTreeStart(*vec);

}

// funkce pro osetreni chyb
static void HandleError( hipError_t error, const char *file, int line ) {
  if (error != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( error ), file, line );
	scanf(" ");
    exit( EXIT_FAILURE );
  }
}
#define BUFFER_EXTENSION 1.5f
#define CHECK_ERROR( error ) ( HandleError( error, __FILE__, __LINE__ ) )

/**
Initialize cuda buffers
*/
void cudaInitialize(int rayBufferSize,int pixCount)
{
	CHECK_ERROR( hipMalloc( (void**)&(CUDA_Nodes), GLOBAL_COMPACT.countNode*sizeof(KDTreeNodeCompact) ) );
	CHECK_ERROR( hipMalloc( (void**)&(CUDA_Spheres), GLOBAL_COMPACT.countSph*sizeof(SphereCompact) ) );
	CHECK_ERROR( hipMalloc( (void**)&(CUDA_Triangles), GLOBAL_COMPACT.countTri*sizeof(TriangleCompact) ) );
	CHECK_ERROR( hipMalloc( (void**)&(CUDA_Materials), GLOBAL_COMPACT.countMat*sizeof(MaterialCompact) ) );
	CHECK_ERROR( hipMalloc( (void**)&(CUDA_Lights), GLOBAL_COMPACT.countPLi*sizeof(PointLightCompact) ) );
	
	CHECK_ERROR( hipMalloc( (void**)&(CUDA_Indices_Tri), GLOBAL_COMPACT.triangleIndicesSize*sizeof(int) ) );
	CHECK_ERROR( hipMalloc( (void**)&(CUDA_Indices_Sph), GLOBAL_COMPACT.sphereIndicesSize*sizeof(int) ) );

	CHECK_ERROR( hipMalloc( (void**)&(CUDA_Rays), rayBufferSize*sizeof(RayCompact) ) );
	CHECK_ERROR( hipMalloc( (void**)&(CUDA_ColorBuffer), pixCount*3*sizeof(float) ) );
}
/**
Free cuda buffers
*/
void hipFree()
{
	hipFree(CUDA_Nodes);
	hipFree(CUDA_Spheres);
	hipFree(CUDA_Triangles);
	hipFree(CUDA_Materials);
	hipFree(CUDA_Lights);
	hipFree(CUDA_Indices_Tri);
	hipFree(CUDA_Indices_Sph);
	
	hipFree(CUDA_Rays);
	hipFree(CUDA_ColorBuffer);
}

/**
Compute length of vector
*/
__device__ float length(float *vec)
{
	return sqrt(vec[0]*vec[0] + vec[1]*vec[1] + vec[2]*vec[2]);
}
/**
Normalize vector
*/
__device__ void normalize(float *vec)
{
	float f = 1.0f / length(vec);
	vec[0] *= f;
	vec[1] *= f;
	vec[2] *= f;
}
/**
Compute dot product
*/
__device__ float dot(float *v0, float *v1)
{
	return v0[0]*v1[0] + v0[1]*v1[1] + v0[2]*v1[2];
}
/**
Compute cross product
*/
__device__ void cross(float *in1, float *in2, float *out)
{
		out[0] = in1[1] * in2[2] - in1[2] * in2[1];
		out[1] = in1[2] * in2[0] - in1[0] * in2[2];
		out[2] = in1[0] * in2[1] - in1[1] * in2[0];
}
/**
Check if point is inside node nodes[nodeId]

/param[in] nodes compacted KD tree
/param[in] nodeId index of node from KD tree
/param[in] point point 
/return true if point is ni node
*/
__device__ bool isInNode(KDTreeNodeCompact *nodes, int nodeId, float *point)
{
	if(nodes[nodeId].bounds[0] > point[0] || nodes[nodeId].bounds[1] < point[0]){return false;}
	if(nodes[nodeId].bounds[2] > point[1] || nodes[nodeId].bounds[3] < point[1]){return false;}
	if(nodes[nodeId].bounds[4] > point[2] || nodes[nodeId].bounds[5] < point[2]){return false;}
	return true;
}
/**
Find point where ray enters node of KDTree.

/param[in] nodes compacted KD tree
/param[in] nodeId index of node from KD tree
/param[in] start origin of ray
/param[in] dir direction of ray (normalized)
/param[in] outPoint output point
/return value depends on entering side(0->-x, 1->x, 2->-y, 3->y, 4->-z, 5->z)
*/
__device__ int findEnterPoint(KDTreeNodeCompact *nodes, int nodeId, float *start, float *dir, float *outPoint)
{
	float corner[3];
	float bounds[6];
	float n[3];
	float nX, nY, nZ, d, u, div, t;
	
	bounds[0] = nodes[nodeId].bounds[0];
	bounds[1] = nodes[nodeId].bounds[1];
	bounds[2] = nodes[nodeId].bounds[2];
	bounds[3] = nodes[nodeId].bounds[3];
	bounds[4] = nodes[nodeId].bounds[4];
	bounds[5] = nodes[nodeId].bounds[5];

	if(dir[0] < 0.0f)
	{
		corner[0] = bounds[1];
		nX = 1;
	}else{
		corner[0] = bounds[0];
		nX = -1;
	}
	if(dir[1] < 0.0f)
	{
		corner[1] = bounds[3];
		nY = 1;
	}else{
		corner[1] = bounds[2];
		nY = -1;
	}
	if(dir[2] < 0.0f)
	{
		corner[2] = bounds[5];
		nZ = 1;
	}else{
		corner[2] = bounds[4];
		nZ = -1;
	}

	//X plane
	n[0] = nX;
	n[1] = 0.0f;
	n[2] = 0.0f;

	d = dot(n, corner);
	u = dot(n, start);
	div = dot(n, dir);

	if(div != 0.0f)
	{
		t = (d-u)/div;
		
		outPoint[0] = start[0] + t * dir[0];
		outPoint[1] = start[1] + t * dir[1];
		outPoint[2] = start[2] + t * dir[2];
		if(outPoint[1] >= bounds[2] && outPoint[1] <= bounds[3] && outPoint[2] >= bounds[4] && outPoint[2] <= bounds[5])
		{
			return (nX < 0)?0:1;
		}
	}

	//Y plane
	n[0] = 0.0f;
	n[1] = nY;
	n[2] = 0.0f;

	d = dot(n, corner);
	u = dot(n, start);
	div = dot(n, dir);

	if(div != 0.0f)
	{
		t = (d-u)/div;
		
		outPoint[0] = start[0] + t * dir[0];
		outPoint[1] = start[1] + t * dir[1];
		outPoint[2] = start[2] + t * dir[2];
		if(outPoint[0] >= bounds[0] && outPoint[0] <= bounds[1] && outPoint[2] >= bounds[4] && outPoint[2] <= bounds[5])
		{
			return (nY < 0)?2:3;
		}
	}

	//Z plane
	n[0] = 0.0f;
	n[1] = 0.0f;
	n[2] = nZ;

	d = dot(n, corner);
	u = dot(n, start);
	div = dot(n, dir);

	if(div != 0.0f)
	{
		t = (d-u)/div;
		
		outPoint[0] = start[0] + t * dir[0];
		outPoint[1] = start[1] + t * dir[1];
		outPoint[2] = start[2] + t * dir[2];
		if(outPoint[1] >= bounds[2] && outPoint[1] <= bounds[3] && outPoint[0] >= bounds[0] && outPoint[0] <= bounds[1])
		{
			return (nZ < 0)?4:5;
		}
	}
	outPoint[0] = 0.0f;
	outPoint[1] = 0.0f;
	outPoint[2] = 0.0f;
	return -1;
}
/**
Find point where ray escapes node of KDTree.

/param[in] nodes compacted KD tree
/param[in] nodeId index of node from KD tree
/param[in] start origin of ray
/param[in] dir direction of ray (normalized)
/param[in] outPoint output point
/return value depends on exiting side(0->-x, 1->x, 2->-y, 3->y, 4->-z, 5->z)
*/
__device__ int findExitPoint(KDTreeNodeCompact *nodes, int nodeId, float *start, float *dir, float *outPoint)
{
	float corner[3];
	float bounds[6];
	float n[3];
	float nX, nY, nZ, d, u, div, t;
	
	bounds[0] = nodes[nodeId].bounds[0];
	bounds[1] = nodes[nodeId].bounds[1];
	bounds[2] = nodes[nodeId].bounds[2];
	bounds[3] = nodes[nodeId].bounds[3];
	bounds[4] = nodes[nodeId].bounds[4];
	bounds[5] = nodes[nodeId].bounds[5];

	if(dir[0] < 0.0f)
	{
		corner[0] = bounds[0];
		nX = 1;
	}else{
		corner[0] = bounds[1];
		nX = -1;
	}
	if(dir[1] < 0.0f)
	{
		corner[1] = bounds[2];
		nY = 1;
	}else{
		corner[1] = bounds[3];
		nY = -1;
	}
	if(dir[2] < 0.0f)
	{
		corner[2] = bounds[4];
		nZ = 1;
	}else{
		corner[2] = bounds[5];
		nZ = -1;
	}

	//X plane
	n[0] = nX;
	n[1] = 0.0f;
	n[2] = 0.0f;

	d = dot(n, corner);
	u = dot(n, start);
	div = dot(n, dir);

	if(div != 0.0f)
	{
		t = (d-u)/div;
		
		outPoint[0] = start[0] + t * dir[0];
		outPoint[1] = start[1] + t * dir[1];
		outPoint[2] = start[2] + t * dir[2];
		if(outPoint[1] >= bounds[2] && outPoint[1] <= bounds[3] && outPoint[2] >= bounds[4] && outPoint[2] <= bounds[5])
		{
			return (nX > 0)?0:1;
		}
	}

	//Y plane
	n[0] = 0.0f;
	n[1] = nY;
	n[2] = 0.0f;

	d = dot(n, corner);
	u = dot(n, start);
	div = dot(n, dir);

	if(div != 0.0f)
	{
		t = (d-u)/div;
		
		outPoint[0] = start[0] + t * dir[0];
		outPoint[1] = start[1] + t * dir[1];
		outPoint[2] = start[2] + t * dir[2];
		if(outPoint[0] >= bounds[0] && outPoint[0] <= bounds[1] && outPoint[2] >= bounds[4] && outPoint[2] <= bounds[5])
		{
			return (nY > 0)?2:3;
		}
	}

	//Z plane
	n[0] = 0.0f;
	n[1] = 0.0f;
	n[2] = nZ;

	d = dot(n, corner);
	u = dot(n, start);
	div = dot(n, dir);

	if(div != 0.0f)
	{
		t = (d-u)/div;
		
		outPoint[0] = start[0] + t * dir[0];
		outPoint[1] = start[1] + t * dir[1];
		outPoint[2] = start[2] + t * dir[2];
		if(outPoint[1] >= bounds[2] && outPoint[1] <= bounds[3] && outPoint[0] >= bounds[0] && outPoint[0] <= bounds[1])
		{
			return (nZ > 0)?4:5;
		}
	}
	outPoint[0] = 0.0f;
	outPoint[1] = 0.0f;
	outPoint[2] = 0.0f;
	return -1;
}
/**
Copy values of one ray to another

/param[in] in ray to copy from
/param[in] out ray to copy to
*/
__device__ void copyRay(RayCompact& in, RayCompact& out)
{
			out.start[0] = in.start[0];
			out.start[1] = in.start[1];
			out.start[2] = in.start[2];
			out.end[0] = in.end[0];
			out.end[1] = in.end[1];
			out.end[2] = in.end[2];
			out.color[0] = in.color[0];
			out.color[1] = in.color[1];
			out.color[2] = in.color[2];
			out.contribution[0] = in.contribution[0];
			out.contribution[1] = in.contribution[1];
			out.contribution[2] = in.contribution[2];
			
			out.length = in.length;

			out.depth = in.depth;
			out.offset = in.offset;
			out.orientation = in.orientation;
			out.type = in.type;
}
/**
Collide sphere with ray

/param[in] ray ray to collide
/param[in] spheres buffer of spheres
/param[in] index index of sphere from spheres
/param[in] impact output point where collision happened
/param[in] normal output vector, normal at impact point
/param[in] len max distance where collision can happen( max length of ray )
/param[in] bound max distance where collision can happen( end of KD tree box )
/return true if collided
*/
__device__ bool collideSphere(RayCompact& ray, SphereCompact *spheres, int index, float *impact, float *normal, float& len, float bound)
{
	float b, c, D;
	float dir[3];
	float temp[3];
	float sphP[3];
	
	sphP[0] = spheres[index].pos[0];
	sphP[1] = spheres[index].pos[1];
	sphP[2] = spheres[index].pos[2];
	
	dir[0] = ray.end[0] - ray.start[0];
	dir[1] = ray.end[1] - ray.start[1];
	dir[2] = ray.end[2] - ray.start[2];
	normalize(dir);
	
	temp[0] = ray.start[0] - sphP[0];
	temp[1] = ray.start[1] - sphP[1];
	temp[2] = ray.start[2] - sphP[2];

	float r = spheres[index].r;
	

	b = 2* dot(dir, temp);
	c = dot(temp, temp) - r*r; 
	D = b*b - 4*c;


	// D < 0
	if(D < 0.0f){return false;}
	// D = 0 
	if(D == 0.0f){return false;}
	// D > 0

	float Ds, t;
	float imp[3];
	float norm[3];
	Ds = sqrt(D);

	if(ray.orientation > 0.0f)
	{
		t = 0.5f * (-b - Ds);
	}else{
		t = 0.5f * (-b + Ds);
	}

	if(t < 0.0f || t > ray.length || t > bound){return false;}

	imp[0] = ray.start[0] + t * dir[0];
	imp[1] = ray.start[1] + t * dir[1];
	imp[2] = ray.start[2] + t * dir[2];
	
	norm[0] = imp[0] - sphP[0];
	norm[1] = imp[1] - sphP[1];
	norm[2] = imp[2] - sphP[2];
	normalize(norm);

	if(ray.orientation * dot(dir, norm) > 0.0f){return false;}
	
	impact[0] = imp[0];
	impact[1] = imp[1];
	impact[2] = imp[2];
	
	normal[0] = norm[0];
	normal[1] = norm[1];
	normal[2] = norm[2];
	len = t;
	return true;
}
/**
Collide ray with triangle

/param[in] ray ray to collide
/param[in] triangles buffer of triangles
/param[in] index index of triangle from triangles
/param[in] impact output point where collision happened
/param[in] normal output vector, normal at impact point
/param[in] len max distance where collision can happen( max length of ray )
/param[in] bound max distance where collision can happen( end of KD tree box )
/return true if collided
*/
__device__ bool collideTriangle(RayCompact& ray, TriangleCompact *triangles, int index, float *impact, float *normal, float& len, float bound)
{
	float v0[3];
	float v1[3];
	float v2[3];
	
	v0[0] = triangles[index].v0[0];
	v0[1] = triangles[index].v0[1];
	v0[2] = triangles[index].v0[2];

	v1[0] = triangles[index].v1[0];
	v1[1] = triangles[index].v1[1];
	v1[2] = triangles[index].v1[2];

	v2[0] = triangles[index].v2[0];
	v2[1] = triangles[index].v2[1];
	v2[2] = triangles[index].v2[2];


	float e1[3];
	float e2[3];
	float pvec[3];
	float tvec[3];
	float qvec[3];
	float dir[3];
	
	e1[0] = v1[0] - v0[0];
	e1[1] = v1[1] - v0[1];
	e1[2] = v1[2] - v0[2];

	e2[0] = v2[0] - v0[0];
	e2[1] = v2[1] - v0[1];
	e2[2] = v2[2] - v0[2];
	
	dir[0] = ray.end[0] - ray.start[0];
	dir[1] = ray.end[1] - ray.start[1];
	dir[2] = ray.end[2] - ray.start[2];
	normalize(dir);

	cross(dir, e2, pvec);
	float det = dot(e1, pvec);

	if(det > -EPSILON && det < EPSILON){return false;}
	
	tvec[0] = ray.start[0] - v0[0];
	tvec[1] = ray.start[1] - v0[1];
	tvec[2] = ray.start[2] - v0[2];

	float u = dot(tvec, pvec);
	if(u < 0.0f || u > det){return false;}

	cross(tvec, e1, qvec);
	float v = dot(dir, qvec);
	if(v < 0.0f || u + v > det){return false;}

	float t = dot(e2, qvec);
	det = 1/det;
	u *= det;
	v *= det;
	t *= det;
	
	if(t < 0.0f || t > len){return false;}
	if(t > bound){return false;}
	
	impact[0] = (1-u-v) * v0[0] + u * v1[0] + v * v2[0];
	impact[1] = (1-u-v) * v0[1] + u * v1[1] + v * v2[1];
	impact[2] = (1-u-v) * v0[2] + u * v1[2] + v * v2[2];
	
	normal[0] = (1-u-v) * triangles[index].n0[0] + u * triangles[index].n1[0] + v * triangles[index].n2[0];
	normal[1] = (1-u-v) * triangles[index].n0[1] + u * triangles[index].n1[1] + v * triangles[index].n2[1];
	normal[2] = (1-u-v) * triangles[index].n0[2] + u * triangles[index].n1[2] + v * triangles[index].n2[2];
	len = t;
	return true;
}
/**
Clip float to values from 0.0f to 1.0f
*/
__device__ float clip(float d)
{
	if(d > 1.0f){return 1.0f;}
	if(d < 0.0f){return 0.0f;}
	return d;
}
/**
Reflect vector with normal

/param[in] dir vector to reflect (normalized)
/param[in] norm normal of reflecting plane
/param[in] out output vector
*/
__device__ void reflect(float *dir, float *norm, float *out)
{
	float mult = -2 * (dir[0] * norm[0] + dir[1] * norm[1] + dir[2] * norm[2]);
	out[0] = dir[0] + norm[0] * mult;
	out[1] = dir[1] + norm[1] * mult;
	out[2] = dir[2] + norm[2] * mult;
}
/**
Compute specular decrease coeficient (clipped dot product of vector to light and ray direction reflected with plane normal )

/param[in] dir ray direction (normalized)
/param[in] toLight vector from reflection point to light
/param[in] normal of reflecting plane
*/
__device__ float getSpecularDecrease(float* dir, float *toLight, float *normal)
{
	float reflected[3];
	reflect(dir, normal, reflected);
	float d = reflected[0] * toLight[0] + reflected[1] * toLight[1] + reflected[2] * toLight[2];

	if( d < 0.0f)	{return 0.0f;}
	if(d > 1.0f)	{return 1.0f;}
	return d;
}
/**
Find leaf there point belongs in tree.

/param[in] nodes compacted KD tree
/param[in] nodeId id of node where to start
/param[in] point point used in search for leaf
/return index of leaf in nodes 
*/
__device__ int findNodeByPoint(KDTreeNodeCompact *nodes, int nodeId, float *point)
{
	if(nodes[nodeId].isLeaf)
	{
		return nodeId;
	}

	int next;
	float split = nodes[nodeId].split;

	if(point[nodes[nodeId].axis] < split)
	{
		next = nodes[nodeId].idLeft; 
	}else{
		next = nodes[nodeId].idRight;
	}
	return findNodeByPoint(nodes, next, point);
}

/**
Trace ray through scene, using basic KDTree traversing algorithm (always search for new leaf from root)

/param[in] ray ray to be traced
/param[in] nodes compacted KD tree
/param[in] spheres sphere buffer
/param[in] triangles triangle buffer
/param[in] materials material buffer
/param[in] lights light buffer
/param[in] lightCount how many lights are there
/param[in] indicesTri buffer of triangle indices
/param[in] indicesSph buffer of sphere indices
/param[in] colorBuffer output color buffer
/param[in] rays global buffer of rays
/param[in] stackInfo index where ray buffer has first free space
/param[in] goDeeper output info, true if reflection happened and ray was rewritten to new reflected ray

*/
__device__ void executeBasic(RayCompact& ray,KDTreeNodeCompact *nodes,SphereCompact *spheres,TriangleCompact *triangles,MaterialCompact *materials,	PointLightCompact *lights, int lightCount,	int *indicesTri,int *indicesSph,float *colorBuffer,	RayCompact *rays,unsigned int& stackInfo, bool& goDeeper)
{
	
	float dir[3];
	float pointIn[3];
	float pointOut[3];
	
	float impact[3];
	float normal[3];

	float dist[3];
	
	int initTri;
	int countTri;
	int initSph;
	int countSph;

	int indexNodeCurrent;
	int indexMat;
	int indexLight;
	int indexObject;

	int found;
	
	bool collided = false;
	bool collidedNow;

	bool refRev = false;
	bool refRevNow;

	float len = ray.length;
	float boxBoundLength;

	float colorMod[3];
	float transp;

	//-------------------------------------------------------------------------------------------------------------

	//compute direction of ray and starting point point
	dir[0] = ray.end[0] - ray.start[0];
	dir[1] = ray.end[1] - ray.start[1];
	dir[2] = ray.end[2] - ray.start[2];
	normalize(dir);

	pointIn[0] = ray.start[0];
	pointIn[1] = ray.start[1];
	pointIn[2] = ray.start[2];

	//skip this if ray is too deep
	if(ray.depth <= MAX_DEPTH)
	{
		//if ray origin isnt in kd tree, find point where it enters
		if(	!isInNode(nodes, 0, pointIn) )
		{
			found = findEnterPoint(nodes,0,ray.start,dir,pointIn);
			pointIn[0] += dir[0] * STEP_SIZE;
			pointIn[1] += dir[1] * STEP_SIZE;
			pointIn[2] += dir[2] * STEP_SIZE;
			if(found == -1)
			{
				pointIn[0] = ray.start[0];
				pointIn[1] = ray.start[1];
				pointIn[2] = ray.start[2];
			}
		}

		//ray traversing routine, first check if point is still in tree
		while( isInNode(nodes, 0, pointIn) )
		{
			indexNodeCurrent = findNodeByPoint(nodes, 0, pointIn);
			//find point, where ray eventuali leaves
			found = findExitPoint(nodes, indexNodeCurrent, ray.start, dir, pointOut);
			//make sure the point is not on border of leaves
			pointOut[0] += dir[0] * STEP_SIZE;
			pointOut[1] += dir[1] * STEP_SIZE;
			pointOut[2] += dir[2] * STEP_SIZE;

			dist[0] = pointOut[0] - ray.start[0];
			dist[1] = pointOut[1] - ray.start[1];
			dist[2] = pointOut[2] - ray.start[2];

			//compute at what length the ray leaves the leaf
			boxBoundLength = length(dist);

			//sphere collisions
			initSph = nodes[indexNodeCurrent].offsetSph;
			countSph = nodes[indexNodeCurrent].countSph;
			for(int i = initSph; i < initSph + countSph; i++)
			{
				indexObject = indicesSph[i];
				//try to collide
				collidedNow = collideSphere(ray, spheres, indexObject, impact, normal, len, boxBoundLength);
				refRevNow = true;
				//check if collision happened in this iteration
				if(collidedNow)
				{
					indexMat = spheres[indexObject].mat;
					transp = materials[indexMat].transparency;
					// if it is shadow ray hiting transparent object, just modify its color, but dont make true collision.
					if( ray.type == RAY_SHADOW && transp < 1.0f)
					{
						ray.color[0] *= (materials[indexMat].diff[0] * materials[indexMat].kD[0] * (1 - transp));
						ray.color[1] *= (materials[indexMat].diff[1] * materials[indexMat].kD[1] * (1 - transp));
						ray.color[2] *= (materials[indexMat].diff[2] * materials[indexMat].kD[2] * (1 - transp)); 
					}else{
						// ray really ends here (if no better collision exists)
						collided = true;
						ray.length = len;
						refRev = refRevNow;
					}
				}
			}

			//triangle collisions
			initTri = nodes[indexNodeCurrent].offsetTri;
			countTri = nodes[indexNodeCurrent].countTri;
			for(int i = initTri; i < initTri + countTri; i++)
			{
				indexObject = indicesTri[i];
				//try to collide
				collidedNow = collideTriangle(ray, triangles, indexObject, impact, normal, len, boxBoundLength);
				refRevNow = false;
				
				//check if collision happened in this iteration
				if(collidedNow)
				{
					indexMat = triangles[indexObject].mat;
					transp = materials[indexMat].transparency;
					// if it is shadow ray hiting transparent object, just modify its color, but dont make true collision.
					if( ray.type == RAY_SHADOW && transp < 1.0f)
					{
						ray.color[0] *= (materials[indexMat].diff[0] * materials[indexMat].kD[0] * transp);
						ray.color[1] *= (materials[indexMat].diff[1] * materials[indexMat].kD[1] * transp);
						ray.color[2] *= (materials[indexMat].diff[2] * materials[indexMat].kD[2] * transp); 
					}else{
						// ray really ends here (if no better collision exists)
						collided = true;
						ray.length = len;
						refRev = refRevNow;
					}
				}
			}

			// if at least one true collision happened, dont traverse tree any further
			if(collided){break;}
			if(found != -1)
			{
				// set new start point
				pointIn[0] = pointOut[0];
				pointIn[1] = pointOut[1];
				pointIn[2] = pointOut[2];
			}else{
				// we are outside tree space
				break;
			}
		}
	}

	if(!collided)
	{
		//no collision happened-> shadow ray reached the light or left scene, in any case just write its color into color buffer 
		float finalCol[3];
		
		finalCol[0] = clip(ray.color[0] * ray.contribution[0]);
		finalCol[1] = clip(ray.color[1] * ray.contribution[1]);
		finalCol[2] = clip(ray.color[2] * ray.contribution[2]);

		int offset = 3 * ray.offset;
		
		atomicAdd(colorBuffer + offset		, finalCol[0]);
		atomicAdd(colorBuffer + offset + 1	, finalCol[1]);
		atomicAdd(colorBuffer + offset + 2	, finalCol[2]);
		return;
	}

	// shadow rays dnt make any reflection, refraction or shadow rays
	if(ray.type == RAY_SHADOW){return;}
	
	RayCompact recursiveRay;
	float vDir[3];
	float atten;
	float specularDecrease;
	float d;
	if(ray.depth < MAX_DEPTH)
	{
		//SHADOW
		if(ray.orientation > 0.0f)
		{
			//go through all lights
			for(int i = 0; i < lightCount; i++)
			{
				//build new ray
				recursiveRay.start[0] = impact[0];
				recursiveRay.start[1] = impact[1];
				recursiveRay.start[2] = impact[2];
				recursiveRay.end[0] = lights[i].pos[0];
				recursiveRay.end[1] = lights[i].pos[1];
				recursiveRay.end[2] = lights[i].pos[2];
				vDir[0] = recursiveRay.end[0] - recursiveRay.start[0];
				vDir[1] = recursiveRay.end[1] - recursiveRay.start[1];
				vDir[2] = recursiveRay.end[2] - recursiveRay.start[2];

				recursiveRay.length = length(vDir);
				normalize(vDir);
				recursiveRay.start[0] += vDir[0] * STEP_SIZE;
				recursiveRay.start[1] += vDir[1] * STEP_SIZE;
				recursiveRay.start[2] += vDir[2] * STEP_SIZE;
				atten = lights[i].intensity/ ((lights[i].a2 * recursiveRay.length + lights[i].a1) * recursiveRay.length + lights[i].a0);
			
				specularDecrease = pow(getSpecularDecrease(dir, vDir, normal), materials[indexMat].shine);
				recursiveRay.color[0] = ( lights[i].col[0] * materials[indexMat].diff[0] + materials[indexMat].kS[0] * specularDecrease) * atten;
				recursiveRay.color[1] = ( lights[i].col[1] * materials[indexMat].diff[1] + materials[indexMat].kS[1] * specularDecrease) * atten;
				recursiveRay.color[2] = ( lights[i].col[2] * materials[indexMat].diff[2] + materials[indexMat].kS[2] * specularDecrease) * atten;
				
				recursiveRay.depth = ray.depth;
				recursiveRay.offset = ray.offset;
				recursiveRay.orientation = 1.0f;
				recursiveRay.type = RAY_SHADOW;
				
				d = clip(dot(vDir, normal)) * materials[indexMat].transparency;
				
				recursiveRay.contribution[0] = ray.contribution[0] * d * materials[indexMat].kD[0];
				recursiveRay.contribution[1] = ray.contribution[1] * d * materials[indexMat].kD[1];
				recursiveRay.contribution[2] = ray.contribution[2] * d * materials[indexMat].kD[2];
				
				bool no = false;
				//recursively execute rhadow ray, it wont go any deeper
				executeBasic(recursiveRay, nodes, spheres, triangles, materials, lights, lightCount, indicesTri, indicesSph, colorBuffer, rays, stackInfo, no);
			}
		}

		//REFRACTED (only in case of transparent objects)
		bool TIR = false;
		if(transp < 1.0f)
		{
			float dotP = dot(dir,normal);
			float tmpNorm[3];
			float ref;
			tmpNorm[0] = normal[0];
			tmpNorm[1] = normal[1];
			tmpNorm[2] = normal[2];

			//am i entering or leaving object<
			if(dotP < 0)
			{
				ref = 1.0f/materials[indexMat].refraction;
			}else{
				ref = materials[indexMat].refraction;
				dotP = -dotP;
				tmpNorm[0] = -tmpNorm[0];
				tmpNorm[1] = -tmpNorm[1];
				tmpNorm[2] = -tmpNorm[2];
			}

			//computation of how much the angle changes
			float s2 = (ref*ref)*(1-(dotP*dotP));
			float sqr = 1 - s2;
			if(sqr < 0.0f)
			{
				//all light will be reflected
				TIR = true;
			}else{
				sqr = -ref * dotP - sqrt(sqr);
				
				vDir[0] = dir[0] * ref + sqr * tmpNorm[0];
				vDir[1] = dir[1] * ref + sqr * tmpNorm[1];
				vDir[2] = dir[2] * ref + sqr * tmpNorm[2];
				
				// allocate one position at end of ray buffer and fill it with refraction ray
				unsigned int bufferAlloc = atomicInc(&stackInfo, UINT_MAX);
				rays[bufferAlloc].start[0] = impact[0] + STEP_SIZE * vDir[0];
				rays[bufferAlloc].start[1] = impact[1] + STEP_SIZE * vDir[1];
				rays[bufferAlloc].start[2] = impact[2] + STEP_SIZE * vDir[2];
				rays[bufferAlloc].end[0] = impact[0] + vDir[0];
				rays[bufferAlloc].end[1] = impact[1] + vDir[1];
				rays[bufferAlloc].end[2] = impact[2] + vDir[2];
				rays[bufferAlloc].color[0] = DEFAULT_COL_R;
				rays[bufferAlloc].color[1] = DEFAULT_COL_G;
				rays[bufferAlloc].color[2] = DEFAULT_COL_B;
				rays[bufferAlloc].contribution[0] = ray.contribution[0];
				rays[bufferAlloc].contribution[1] = ray.contribution[1];
				rays[bufferAlloc].contribution[2] = ray.contribution[2];

				rays[bufferAlloc].length = MAX_DIST;

				rays[bufferAlloc].depth = ray.depth + 1;
				rays[bufferAlloc].offset = ray.offset;
				rays[bufferAlloc].orientation = ((refRev) ? -1.0f : 1.0f) * ray.orientation;
				rays[bufferAlloc].type = RAY_REFRACTION;
			}
		}
	
		//REFLECTED (only in case of reflective objects)
		float refl = materials[indexMat].reflection;
		if(refl > 0.0f)
		{
			//build reflection ray
			//because this is the ond of this function, it is pointless to trace this ray recursively
			//it will be better to finish this, rewrite original ray and say the caller, that he should repeat the process
			float reflected[3];
			reflect(dir, normal, reflected);

			
			ray.start[0]	= impact[0] + STEP_SIZE * reflected[0];
			ray.start[1]	= impact[1] + STEP_SIZE * reflected[1];
			ray.start[2]	= impact[2] + STEP_SIZE * reflected[2];
			ray.end[0]		= impact[0] + reflected[0];
			ray.end[1]		= impact[1] + reflected[1];
			ray.end[2]		= impact[2] + reflected[2];

			ray.color[0]	= DEFAULT_COL_R;
			ray.color[1]	= DEFAULT_COL_G;
			ray.color[2]	= DEFAULT_COL_B;

			ray.contribution[0]	= ray.contribution[0] * materials[indexMat].kS[0] * refl * ((TIR)?1.0f:transp);
			ray.contribution[1]	= ray.contribution[1] * materials[indexMat].kS[1] * refl * ((TIR)?1.0f:transp);
			ray.contribution[2]	= ray.contribution[2] * materials[indexMat].kS[2] * refl * ((TIR)?1.0f:transp);

			ray.length = MAX_DIST;

			ray.depth = ray.depth + 1;
			ray.offset = ray.offset;
			ray.orientation = 1.0f;
			ray.type = RAY_REFLECTION;
			//this sayt it should be repeated
			goDeeper = true;
		}

	}
}
/**
Trace ray through scene, using neighborhood KDTree traversing(Each node knows what node is next to them so search for next node doesnt have to start from root)
For commented code go to _device_ void executeBasic(...) it is nearly identical

/param[in] ray ray to be traced
/param[in] nodes compacted KD tree
/param[in] spheres sphere buffer
/param[in] triangles triangle buffer
/param[in] materials material buffer
/param[in] lights light buffer
/param[in] lightCount how many lights are there
/param[in] indicesTri buffer of triangle indices
/param[in] indicesSph buffer of sphere indices
/param[in] colorBuffer output color buffer
/param[in] rays global buffer of rays
/param[in] stackInfo index where ray buffer has first free space
/param[in] goDeeper output info, true if reflection happened and ray was rewritten to new reflected ray
*/
__device__ void executeNeighbor(RayCompact& ray,KDTreeNodeCompact *nodes,SphereCompact *spheres,TriangleCompact *triangles,MaterialCompact *materials,	PointLightCompact *lights, int lightCount,	int *indicesTri,int *indicesSph,float *colorBuffer,	RayCompact *rays,unsigned int& stackInfo, bool& goDeeper)
{
	float dir[3];
	float pointIn[3];
	float pointOut[3];
	
	float impact[3];
	float normal[3];

	float dist[3];
	
	int initTri;
	int countTri;
	int initSph;
	int countSph;

	int indexNodeCurrent;
	int indexMat;
	int indexLight;
	int indexObject;

	int indexSubTree;

	int found;
	
	bool collided = false;
	bool collidedNow;

	bool refRev = false;
	bool refRevNow;

	float len = ray.length;
	float boxBoundLength;

	float colorMod[3];
	float transp;

	//-------------------------------------------------------------------------------------------------------------

	dir[0] = ray.end[0] - ray.start[0];
	dir[1] = ray.end[1] - ray.start[1];
	dir[2] = ray.end[2] - ray.start[2];
	normalize(dir);

	pointIn[0] = ray.start[0];
	pointIn[1] = ray.start[1];
	pointIn[2] = ray.start[2];

	indexSubTree = 0;

	if(ray.depth <= MAX_DEPTH)
	{
		if(	!isInNode(nodes, 0, pointIn) )
		{
			found = findEnterPoint(nodes,0,ray.start,dir,pointIn);
			pointIn[0] += dir[0] * STEP_SIZE;
			pointIn[1] += dir[1] * STEP_SIZE;
			pointIn[2] += dir[2] * STEP_SIZE;
			if(found == -1)
			{
				pointIn[0] = ray.start[0];
				pointIn[1] = ray.start[1];
				pointIn[2] = ray.start[2];
			}
		}

		while( isInNode(nodes, 0, pointIn) )
		{
			indexNodeCurrent = findNodeByPoint(nodes, indexSubTree, pointIn);

			found = findExitPoint(nodes, indexNodeCurrent, ray.start, dir, pointOut);
			pointOut[0] += dir[0] * STEP_SIZE;
			pointOut[1] += dir[1] * STEP_SIZE;
			pointOut[2] += dir[2] * STEP_SIZE;

			dist[0] = pointOut[0] - ray.start[0];
			dist[1] = pointOut[1] - ray.start[1];
			dist[2] = pointOut[2] - ray.start[2];

			boxBoundLength = length(dist);

			//sphere collisions
			initSph = nodes[indexNodeCurrent].offsetSph;
			countSph = nodes[indexNodeCurrent].countSph;
			for(int i = initSph; i < initSph + countSph; i++)
			{
				indexObject = indicesSph[i];
				collidedNow = collideSphere(ray, spheres, indexObject, impact, normal, len, boxBoundLength);
				refRevNow = true;
				if(collidedNow)
				{
					indexMat = spheres[indexObject].mat;
					transp = materials[indexMat].transparency;
					if( ray.type == RAY_SHADOW && transp < 1.0f)
					{
						ray.color[0] *= (materials[indexMat].diff[0] * materials[indexMat].kD[0] * (1 - transp));
						ray.color[1] *= (materials[indexMat].diff[1] * materials[indexMat].kD[1] * (1 - transp));
						ray.color[2] *= (materials[indexMat].diff[2] * materials[indexMat].kD[2] * (1 - transp)); 
					}else{
						collided = true;
						ray.length = len;
						refRev = refRevNow;
					}
				}
			}

			//triangle collisions
			initTri = nodes[indexNodeCurrent].offsetTri;
			countTri = nodes[indexNodeCurrent].countTri;
			for(int i = initTri; i < initTri + countTri; i++)
			{
				indexObject = indicesTri[i];
				collidedNow = collideTriangle(ray, triangles, indexObject, impact, normal, len, boxBoundLength);
				refRevNow = false;

				if(collidedNow)
				{
					indexMat = triangles[indexObject].mat;
					transp = materials[indexMat].transparency;

					if( ray.type == RAY_SHADOW && transp < 1.0f)
					{
						ray.color[0] *= (materials[indexMat].diff[0] * materials[indexMat].kD[0] * transp);
						ray.color[1] *= (materials[indexMat].diff[1] * materials[indexMat].kD[1] * transp);
						ray.color[2] *= (materials[indexMat].diff[2] * materials[indexMat].kD[2] * transp); 
					}else{
						collided = true;
						ray.length = len;
						refRev = refRevNow;
					}
				}
			}

			if(collided){break;}
			if(found != -1)
			{
				indexSubTree = nodes[indexNodeCurrent].neighbors[found];
				pointIn[0] = pointOut[0];
				pointIn[1] = pointOut[1];
				pointIn[2] = pointOut[2];
			}else{
				break;
			}
		}
	}

	if(!collided)
	{
		float finalCol[3];
		
		finalCol[0] = clip(ray.color[0] * ray.contribution[0]);
		finalCol[1] = clip(ray.color[1] * ray.contribution[1]);
		finalCol[2] = clip(ray.color[2] * ray.contribution[2]);

		int offset = 3 * ray.offset;
		
		atomicAdd(colorBuffer + offset		, finalCol[0]);
		atomicAdd(colorBuffer + offset + 1	, finalCol[1]);
		atomicAdd(colorBuffer + offset + 2	, finalCol[2]);
		return;
	}

	if(ray.type == RAY_SHADOW){return;}
	
	RayCompact recursiveRay;
	float vDir[3];
	float atten;
	float specularDecrease;
	float d;
	if(ray.depth < MAX_DEPTH)
	{
		//SHADOW
		if(ray.orientation > 0.0f)
		{
			for(int i = 0; i < lightCount; i++)
			{
				recursiveRay.start[0] = impact[0];
				recursiveRay.start[1] = impact[1];
				recursiveRay.start[2] = impact[2];
				recursiveRay.end[0] = lights[i].pos[0];
				recursiveRay.end[1] = lights[i].pos[1];
				recursiveRay.end[2] = lights[i].pos[2];
				vDir[0] = recursiveRay.end[0] - recursiveRay.start[0];
				vDir[1] = recursiveRay.end[1] - recursiveRay.start[1];
				vDir[2] = recursiveRay.end[2] - recursiveRay.start[2];
				recursiveRay.length = length(vDir);
				normalize(vDir);
				recursiveRay.start[0] += vDir[0] * STEP_SIZE;
				recursiveRay.start[1] += vDir[1] * STEP_SIZE;
				recursiveRay.start[2] += vDir[2] * STEP_SIZE;
				atten = lights[i].intensity/ ((lights[i].a2 * recursiveRay.length + lights[i].a1) * recursiveRay.length + lights[i].a0);

				specularDecrease = pow(getSpecularDecrease(dir, vDir, normal), materials[indexMat].shine);       
				recursiveRay.color[0] = ( lights[i].col[0] * materials[indexMat].diff[0] + materials[indexMat].kS[0] * specularDecrease) * atten;
				recursiveRay.color[1] = ( lights[i].col[1] * materials[indexMat].diff[1] + materials[indexMat].kS[1] * specularDecrease) * atten;
				recursiveRay.color[2] = ( lights[i].col[2] * materials[indexMat].diff[2] + materials[indexMat].kS[2] * specularDecrease) * atten;
				
				recursiveRay.depth = ray.depth;
				recursiveRay.offset = ray.offset;
				recursiveRay.orientation = 1.0f;
				recursiveRay.type = RAY_SHADOW;
				
				d = clip(dot(vDir, normal)) * materials[indexMat].transparency;
				
				recursiveRay.contribution[0] = ray.contribution[0] * d * materials[indexMat].kD[0];
				recursiveRay.contribution[1] = ray.contribution[1] * d * materials[indexMat].kD[1];
				recursiveRay.contribution[2] = ray.contribution[2] * d * materials[indexMat].kD[2];
				
				bool no = false;
				executeNeighbor(recursiveRay, nodes, spheres, triangles, materials, lights, lightCount, indicesTri, indicesSph, colorBuffer, rays, stackInfo, no);
			}
		}

		//REFRACTED
		bool TIR = false;
		if(transp < 1.0f)
		{
			float dotP = dot(dir,normal);
			float tmpNorm[3];
			float ref;
			tmpNorm[0] = normal[0];
			tmpNorm[1] = normal[1];
			tmpNorm[2] = normal[2];

			if(dotP < 0)
			{
				ref = 1.0f/materials[indexMat].refraction;
			}else{
				ref = materials[indexMat].refraction;
				dotP = -dotP;
				tmpNorm[0] = -tmpNorm[0];
				tmpNorm[1] = -tmpNorm[1];
				tmpNorm[2] = -tmpNorm[2];
			}

			float s2 = (ref*ref)*(1-(dotP*dotP));
			float sqr = 1 - s2;
			if(sqr < 0.0f)
			{
				TIR = true;
			}else{
				sqr = -ref * dotP - sqrt(sqr);
				
				vDir[0] = dir[0] * ref + sqr * tmpNorm[0];
				vDir[1] = dir[1] * ref + sqr * tmpNorm[1];
				vDir[2] = dir[2] * ref + sqr * tmpNorm[2];
				
				unsigned int bufferAlloc = atomicInc(&stackInfo, UINT_MAX);
				rays[bufferAlloc].start[0] = impact[0] + STEP_SIZE * vDir[0];
				rays[bufferAlloc].start[1] = impact[1] + STEP_SIZE * vDir[1];
				rays[bufferAlloc].start[2] = impact[2] + STEP_SIZE * vDir[2];
				rays[bufferAlloc].end[0] = impact[0] + vDir[0];
				rays[bufferAlloc].end[1] = impact[1] + vDir[1];
				rays[bufferAlloc].end[2] = impact[2] + vDir[2];
				rays[bufferAlloc].color[0] = DEFAULT_COL_R;
				rays[bufferAlloc].color[1] = DEFAULT_COL_G;
				rays[bufferAlloc].color[2] = DEFAULT_COL_B;
				rays[bufferAlloc].contribution[0] = ray.contribution[0];
				rays[bufferAlloc].contribution[1] = ray.contribution[1];
				rays[bufferAlloc].contribution[2] = ray.contribution[2];

				rays[bufferAlloc].length = MAX_DIST;

				rays[bufferAlloc].depth = ray.depth + 1;
				rays[bufferAlloc].offset = ray.offset;
				rays[bufferAlloc].orientation = ((refRev) ? -1.0f : 1.0f) * ray.orientation;
				rays[bufferAlloc].type = RAY_REFRACTION;
			}
		}
	
		//REFLECTED
		float refl = materials[indexMat].reflection;
		if(refl > 0.0f)
		{
			float reflected[3];
			reflect(dir, normal, reflected);

			
			ray.start[0]	= impact[0] + STEP_SIZE * reflected[0];
			ray.start[1]	= impact[1] + STEP_SIZE * reflected[1];
			ray.start[2]	= impact[2] + STEP_SIZE * reflected[2];
			ray.end[0]		= impact[0] + reflected[0];
			ray.end[1]		= impact[1] + reflected[1];
			ray.end[2]		= impact[2] + reflected[2];

			ray.color[0]	= DEFAULT_COL_R;
			ray.color[1]	= DEFAULT_COL_G;
			ray.color[2]	= DEFAULT_COL_B;

			ray.contribution[0]	= ray.contribution[0] * materials[indexMat].kS[0] * refl * ((TIR)?1.0f:transp);
			ray.contribution[1]	= ray.contribution[1] * materials[indexMat].kS[1] * refl * ((TIR)?1.0f:transp);
			ray.contribution[2]	= ray.contribution[2] * materials[indexMat].kS[2] * refl * ((TIR)?1.0f:transp);

			ray.length = MAX_DIST;

			ray.depth = ray.depth + 1;
			ray.offset = ray.offset;
			ray.orientation = 1.0f;
			ray.type = RAY_REFLECTION;
			goDeeper = true;
		}
	}
}

/**
Kernel that uses executeBasic (basic KDTree traversing)

/param[in] maxRays max size of ray buffer
/param[in] initRays count of rays at beginning
/param[in] nodes compacted KD tree
/param[in] spheres sphere buffer
/param[in] triangles triangle buffer
/param[in] materials material buffer
/param[in] lights light buffer
/param[in] lightCount how many lights are there
/param[in] indicesTri buffer of triangle indices
/param[in] indicesSph buffer of sphere indices
/param[in] colorBuffer output color buffer
/param[in] rays global buffer of rays
*/
__global__ void kernelGPUBasic(	int maxRays,int initRays,KDTreeNodeCompact *nodes,SphereCompact *spheres,TriangleCompact *triangles,MaterialCompact *materials,PointLightCompact *lights,int lightCount,int *indicesTri,int *indicesSph,float *colorBuffer,	RayCompact *rays)
{

	__shared__ unsigned int STACK_INFO;
	__shared__ unsigned int STACK_NEXT_BATCH_SIZE;

	RayCompact ray_copy;
	SphereCompact sphereC;
	TriangleCompact triangleC;
	int id = threadIdx.x;
	
	// first thread sets shared info (how much rays is in buffer)
	if(id == 0)
	{
		STACK_INFO = initRays;
	}
	__syncthreads();
	int temp;
	int currentRay;
	unsigned int bufferAlloc;

	//main loop
	while(STACK_INFO > 0)
	{
		//first thread bites off part of buffer and tells how many thread is supposed to execute tasks
		if(id == 0)
		{
			temp = STACK_INFO;
			if(temp > blockDim.x)
			{
				//all threads will work
				temp -= blockDim.x;
				STACK_INFO = temp;
				STACK_NEXT_BATCH_SIZE = blockDim.x;
			}else{
				// not enough rays, not all threads will work
				STACK_INFO = 0;
				STACK_NEXT_BATCH_SIZE = temp;
			}
		}
		__syncthreads();

		//reading phase
		if(id < STACK_NEXT_BATCH_SIZE)
		{
			currentRay = STACK_INFO + id;
			copyRay(rays[currentRay],ray_copy);
		}
		
		__syncthreads();

		if(id < STACK_NEXT_BATCH_SIZE)
		{
			//working phase
			bool goAgain = true;
			while(goAgain)
			{
				goAgain = false;
				executeBasic(ray_copy,nodes,spheres,triangles,materials,lights,lightCount,indicesTri,indicesSph,colorBuffer,rays,STACK_INFO,goAgain);
			}
		}
		
		__syncthreads();
	}
}
/**
Kernel that uses executeNeighbor (advanced KDTree traversing)

/param[in] maxRays max size of ray buffer
/param[in] initRays count of rays at beginning
/param[in] nodes compacted KD tree
/param[in] spheres sphere buffer
/param[in] triangles triangle buffer
/param[in] materials material buffer
/param[in] lights light buffer
/param[in] lightCount how many lights are there
/param[in] indicesTri buffer of triangle indices
/param[in] indicesSph buffer of sphere indices
/param[in] colorBuffer output color buffer
/param[in] rays global buffer of rays

for more commented version check kernelGPUBasic, there are just a few changes.
*/
__global__ void kernelGPUNeighbor(	int maxRays,int initRays,KDTreeNodeCompact *nodes,SphereCompact *spheres,TriangleCompact *triangles,MaterialCompact *materials,PointLightCompact *lights,int lightCount,int *indicesTri,int *indicesSph,float *colorBuffer,	RayCompact *rays)
{
	__shared__ unsigned int STACK_INFO;
	__shared__ unsigned int STACK_NEXT_BATCH_SIZE;

	RayCompact ray_copy;
	SphereCompact sphereC;
	TriangleCompact triangleC;
	int id = threadIdx.x;

	// first thread sets shared info
	if(id == 0)
	{
		STACK_INFO = initRays;
	}
	__syncthreads();
	int temp;
	int currentRay;
	unsigned int bufferAlloc;

	//main loop
	while(STACK_INFO > 0)
	{
		//first thread bites off part of buffer and tells how many thread is supposed to execute tasks
		if(id == 0)
		{
			temp = STACK_INFO;
			if(temp > blockDim.x)
			{
				temp -= blockDim.x;
				STACK_INFO = temp;
				STACK_NEXT_BATCH_SIZE = blockDim.x;
			}else{
				STACK_INFO = 0;
				STACK_NEXT_BATCH_SIZE = temp;
			}
		}
		__syncthreads();

		//reading phase
		if(id < STACK_NEXT_BATCH_SIZE)
		{
			currentRay = STACK_INFO + id;
			copyRay(rays[currentRay],ray_copy);
		}
		
		__syncthreads();

		if(id < STACK_NEXT_BATCH_SIZE)
		{
			bool goAgain = true;
			while(goAgain)
			{
				goAgain = false;
				executeNeighbor(ray_copy,nodes,spheres,triangles,materials,lights,lightCount,indicesTri,indicesSph,colorBuffer,rays,STACK_INFO,goAgain);
			}
		}
		
		__syncthreads();
	}
}
/**
Clips float to values from 0.0f to 1.0f
*/
float clip0To1(float f)
{
	if(f > 1.0f){return 1.0f;}
	if(f < 0.0f){return 0.0f;}
	return f;
}
/**
Copy values from GLOBAL_PIXEL_BUFFER to GLOBAL_BITMAP
*/
void pixelBufferToBitmap()
{
	int h = GLOBAL_BITMAP->height;
	int w = GLOBAL_BITMAP->width;
	int offset;
	for(int i = 0; i < h*w; i+=1)
	{
		offset = 3*i;
		GLOBAL_BITMAP->pixels[i].x = 255 * clip0To1(GLOBAL_PIXEL_BUFFER[offset]);
		GLOBAL_BITMAP->pixels[i].y = 255 * clip0To1(GLOBAL_PIXEL_BUFFER[offset + 1]);
		GLOBAL_BITMAP->pixels[i].z = 255 * clip0To1(GLOBAL_PIXEL_BUFFER[offset + 2]);
	}
}
/**
Method handling keyboard events.
*/
static void HandleKeys(unsigned char key, int x, int y) {
  switch (key) {
    case 27:
      //exit(0);
		break;
  }
}

void idleFunc() {
  glutPostRedisplay();
}
/**
Build scene chosen by user
*/
void buildSceneTree()
{
	switch(INPUT_SCENE)
	{
	case 0:
		buildTestScene1();
		break;
	case 1:
		buildTestScene2();
		break;
	case 2:
		buildTestScene3();
		break;
	case 3:
		buildTestScene4();
		break;
	case 4:
		buildTestScene5();
		break;
	case 5:
		buildTestBall();
		break;
	case 6:
		buildMassiveScene1();
		break;
	case 7:
		buildMassiveScene2();
		break;
	case 8:
		buildMassiveScene3();
		break;
	}
}
/**
Prepare and execute raytracing on GPU.
*/
void rayTraceSceneGPU()
{
	
	if(GLOBAL_BITMAP == NULL){printf("bMap not initialized\n"); return;}
	if(GLOBAL_BITMAP->pixels == NULL){printf("pixels not initialized\n"); return;}
	
	int countPixels = GLOBAL_BITMAP->width * GLOBAL_BITMAP->height;
	GLOBAL_PIXEL_BUFFER = new float[countPixels*3];
	int bufferSize = (int)(BUFFER_EXTENSION * countPixels);
	GLOBAL_RAYBUFFER = new RayCompact[bufferSize];

	cam.buildRays(*GLOBAL_BITMAP,GLOBAL_RAYBUFFER,bufferSize);
	
	//cam.buildCentralRay(*GLOBAL_BITMAP,GLOBAL_RAYBUFFER,1);
	//countPixels = 1;

	cudaInitialize(bufferSize, countPixels);

	memset(GLOBAL_PIXEL_BUFFER, 0, countPixels * 3 * sizeof(float));
	
	hipMemcpy(CUDA_Nodes,			GLOBAL_COMPACT.nodes,			GLOBAL_COMPACT.countNode*sizeof(KDTreeNodeCompact), hipMemcpyHostToDevice);
	hipMemcpy(CUDA_Spheres,		GLOBAL_COMPACT.sphere,			GLOBAL_COMPACT.countSph*sizeof(SphereCompact),		hipMemcpyHostToDevice);
	hipMemcpy(CUDA_Triangles,		GLOBAL_COMPACT.tri,				GLOBAL_COMPACT.countTri*sizeof(TriangleCompact),	hipMemcpyHostToDevice);
	hipMemcpy(CUDA_Materials,		GLOBAL_COMPACT.mat,				GLOBAL_COMPACT.countMat*sizeof(MaterialCompact),	hipMemcpyHostToDevice);
	hipMemcpy(CUDA_Lights,			GLOBAL_COMPACT.pLi,				GLOBAL_COMPACT.countPLi*sizeof(PointLightCompact),	hipMemcpyHostToDevice);
	hipMemcpy(CUDA_Indices_Tri,	GLOBAL_COMPACT.trinagleIndices, GLOBAL_COMPACT.triangleIndicesSize*sizeof(int),		hipMemcpyHostToDevice);
	hipMemcpy(CUDA_Indices_Sph,	GLOBAL_COMPACT.sphereIndices,	GLOBAL_COMPACT.sphereIndicesSize*sizeof(int),		hipMemcpyHostToDevice);
	hipMemcpy(CUDA_ColorBuffer,	GLOBAL_PIXEL_BUFFER,			countPixels*3*sizeof(float),						hipMemcpyHostToDevice);
	hipMemcpy(CUDA_Rays,			GLOBAL_RAYBUFFER,				bufferSize*sizeof(RayCompact),						hipMemcpyHostToDevice);
	
	

	dim3 blocks(1, 1);
	dim3 threads(512, 1);
	
	//printCompactTree();
	switch(INPUT_TRAVERSING)
	{
	case 2:
		kernelGPUNeighbor<<<blocks,threads>>>(bufferSize,countPixels,CUDA_Nodes,CUDA_Spheres,CUDA_Triangles,CUDA_Materials,CUDA_Lights,GLOBAL_COMPACT.countPLi,CUDA_Indices_Tri,CUDA_Indices_Sph,CUDA_ColorBuffer,CUDA_Rays);
		break;
	default:
		kernelGPUBasic<<<blocks,threads>>>(bufferSize,countPixels,CUDA_Nodes,CUDA_Spheres,CUDA_Triangles,CUDA_Materials,CUDA_Lights,GLOBAL_COMPACT.countPLi,CUDA_Indices_Tri,CUDA_Indices_Sph,CUDA_ColorBuffer,CUDA_Rays);
		break;
	}
	//ker<<<blocks, threads>>>();
	hipMemcpy(GLOBAL_PIXEL_BUFFER, CUDA_ColorBuffer,				countPixels*3*sizeof(float),						hipMemcpyDeviceToHost);
	pixelBufferToBitmap();
	hipFree();


}
/**
Prepare and execute raytracing on CPU
*/
void raytraceSceneCPU()
{
	if(GLOBAL_BITMAP == NULL){printf("bMap not initialized\n"); return;}
	if(GLOBAL_BITMAP->pixels == NULL){printf("pixels not initialized\n"); return;}

	
	GLOBAL_BITMAP->clear();
	cam.buildRays(*GLOBAL_BITMAP,buffer,GLOBAL_BUFFER_LIGHT);

	RayLink *t;
	t = buffer.start->next;
	switch(INPUT_TRAVERSING)
	{
		case 2:
			while(buffer.size > 0)
			{
				buffer.size--;
				t->task->executeNeighbor(GLOBAL_TREE);
				t = t->next;
			}
			break;
		default:
			while(buffer.size > 0)
			{
				buffer.size--;
				t->task->executeBasic(GLOBAL_TREE);
				t = t->next;
			}
			break;
	}
	printf("drawn\n");
	buffer.clear();
}
/**
Read initial user inputs
*/
void readInputs()
{	
	printf("\nWhich method of rendering? \n0) - CPU \n1) - GPU\n-> ");
	std::cin >> INPUT_METHOD;
	if(INPUT_METHOD < 0 || INPUT_METHOD > 1){exit(1);}
	printf("\nWhich method of traversing? \n0) naive (no KDTree) \n1) - always from root \n2) - next cell\n-> ");
	std::cin >> INPUT_TRAVERSING;
	if(INPUT_TRAVERSING < 0 || INPUT_TRAVERSING > 2){exit(1);}
	if(INPUT_TRAVERSING == 0)
	{
		BUILD_KDTREE = false;
	}
	printf("\nSelect scene (0-8)\n 0 - basic\n 1-insane cornel\n 2-2*2*4 balls\n 3-triangle, just triangle\n 4-nearly purple plane\n 5-basic lens\n 6-ball-o-calypse\n 7-...\n 8-...\n-> ");
	std::cin >> INPUT_SCENE;
	printf("\n");
	if(INPUT_SCENE < 0 || INPUT_SCENE > 8){exit(1);}

}
/**
Main
*/
int main(int argc, char **argv) {
	
	readInputs();

	printf("initializing\n");
  initializeBitmap();
  initializeLight();
  Material::initializeMaterials();
    
  buildSceneTree();
  compactizeCurrentTree();

  printf("\n\n");
  //GLOBAL_TREE.printNeighbors();
  //printCompact();
	printf("begin raytracing\n");
	long long timestampBeforeRender = getTime();
	switch(INPUT_METHOD)
	{
	case 0:
		raytraceSceneCPU();
		break;
	case 1:
		rayTraceSceneGPU();
		break;
	default:
		printf(" %d is not valid method for rendering\n", INPUT_METHOD);
		exit(1);
	}
	long long timestampAfterRender = getTime();
	printf(" rendering took %ld ms\n",(timestampAfterRender- timestampBeforeRender));
	printf("finished raytracing\n");

  //printf("rays - created: %d executed: %d \n", countRayCreated, countRayExecuted);

  glutInit(&argc, argv);

  glutInitWindowSize(IMAGE_WIDTH, IMAGE_HEIGHT);
  glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA );

  glutCreateWindow("RayTracer");
  glutDisplayFunc(DisplayFunc);
  glutKeyboardFunc(HandleKeys);

  glutIdleFunc(idleFunc);

  glutMainLoop();

  return 0;
}